#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2022, imec Vision Lab, University of Antwerp
           2014-2022, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "../../astra/cuda/2d/util.h"
#include "../../astra/cuda/2d/arith.h"

#include <cstdio>
#include <cassert>
#include <iostream>
#include <list>

namespace astraCUDA {

static const unsigned g_MaxAngles = 2560;
__constant__ float gC_SrcX[g_MaxAngles];
__constant__ float gC_SrcY[g_MaxAngles];
__constant__ float gC_DetSX[g_MaxAngles];
__constant__ float gC_DetSY[g_MaxAngles];
__constant__ float gC_DetUX[g_MaxAngles];
__constant__ float gC_DetUY[g_MaxAngles];


// optimization parameters
static const unsigned int g_anglesPerBlock = 16;
static const unsigned int g_detBlockSize = 32;
static const unsigned int g_blockSlices = 64;

// projection for angles that are roughly horizontal
// (detector roughly vertical)
__global__ void FanFPhorizontal(float* D_projData, unsigned int projPitch, hipTextureObject_t tex, unsigned int startSlice, unsigned int startAngle, unsigned int endAngle, const SDimensions dims, float outputScale)
{
	float* projData = (float*)D_projData;
	const int relDet = threadIdx.x;
	const int relAngle = threadIdx.y;

	const int angle = startAngle + blockIdx.x * g_anglesPerBlock + relAngle;
	if (angle >= endAngle)
		return;

	const int detector = blockIdx.y * g_detBlockSize + relDet;

	if (detector < 0 || detector >= dims.iProjDets)
		return;

	const float fSrcX = gC_SrcX[angle];
	const float fSrcY = gC_SrcY[angle];
	const float fDetSX = gC_DetSX[angle];
	const float fDetSY = gC_DetSY[angle];
	const float fDetUX = gC_DetUX[angle];
	const float fDetUY = gC_DetUY[angle];

	float fVal = 0.0f;

	const float fdx = fabsf(fDetSX + detector*fDetUX + 0.5f - fSrcX);
	const float fdy = fabsf(fDetSY + detector*fDetUY + 0.5f - fSrcY);

	if (fdy > fdx)
		return;


	for (int iSubT = 0; iSubT < dims.iRaysPerDet; ++iSubT) {
		const float fDet = detector + (0.5f + iSubT) / dims.iRaysPerDet;

		const float fDetX = fDetSX + fDet * fDetUX;
		const float fDetY = fDetSY + fDet * fDetUY;

		// ray: y = alpha * x + beta
		const float alpha = (fSrcY - fDetY) / (fSrcX - fDetX);
		const float beta = fSrcY - alpha * fSrcX;
	
		const float fDistCorr = sqrt(alpha*alpha+1.0f) * outputScale / dims.iRaysPerDet;

		// intersect ray with first slice

		float fY = -alpha * (startSlice - 0.5f*dims.iVolWidth + 0.5f) - beta + 0.5f*dims.iVolHeight - 0.5f + 0.5f;
		float fX = startSlice + 0.5f;

		int endSlice = startSlice + g_blockSlices;
		if (endSlice > dims.iVolWidth)
			endSlice = dims.iVolWidth;

		float fV = 0.0f;
		for (int slice = startSlice; slice < endSlice; ++slice)
		{
			fV += tex2D<float>(tex, fX, fY);
			fY -= alpha;
			fX += 1.0f;
		}

		fVal += fV * fDistCorr;

	}

	projData[angle*projPitch+detector] += fVal;
}


// projection for angles that are roughly vertical
// (detector roughly horizontal)
__global__ void FanFPvertical(float* D_projData, unsigned int projPitch, hipTextureObject_t tex, unsigned int startSlice, unsigned int startAngle, unsigned int endAngle, const SDimensions dims, float outputScale)
{
	const int relDet = threadIdx.x;
	const int relAngle = threadIdx.y;

	const int angle = startAngle + blockIdx.x * g_anglesPerBlock + relAngle;

	if (angle >= endAngle)
		return;

	const int detector = blockIdx.y * g_detBlockSize + relDet;

	if (detector < 0 || detector >= dims.iProjDets)
		return;

	float* projData = (float*)D_projData;

	const float fSrcX = gC_SrcX[angle];
	const float fSrcY = gC_SrcY[angle];
	const float fDetSX = gC_DetSX[angle];
	const float fDetSY = gC_DetSY[angle];
	const float fDetUX = gC_DetUX[angle];
	const float fDetUY = gC_DetUY[angle];

	float fVal = 0.0f;

	const float fdx = fabsf(fDetSX + detector*fDetUX + 0.5f - fSrcX);
	const float fdy = fabsf(fDetSY + detector*fDetUY + 0.5f - fSrcY);

	if (fdy <= fdx)
		return;


	for (int iSubT = 0; iSubT < dims.iRaysPerDet; ++iSubT) {
		const float fDet = detector + (0.5f + iSubT) / dims.iRaysPerDet /*- gC_angle_offset[angle]*/;

		const float fDetX = fDetSX + fDet * fDetUX;
		const float fDetY = fDetSY + fDet * fDetUY;

		// ray: x = alpha * y + beta
		const float alpha = (fSrcX - fDetX) / (fSrcY - fDetY);
		const float beta = fSrcX - alpha * fSrcY;
	
		const float fDistCorr = sqrt(alpha*alpha+1) * outputScale / dims.iRaysPerDet;

		// intersect ray with first slice

		float fX = -alpha * (startSlice - 0.5f*dims.iVolHeight + 0.5f) + beta + 0.5f*dims.iVolWidth - 0.5f + 0.5f;
		float fY = startSlice + 0.5f;

		int endSlice = startSlice + g_blockSlices;
		if (endSlice > dims.iVolHeight)
			endSlice = dims.iVolHeight;

		float fV = 0.0f;

		for (int slice = startSlice; slice < endSlice; ++slice)
		{
			fV += tex2D<float>(tex, fX, fY);
			fX -= alpha;
			fY += 1.0f;
		}

		fVal += fV * fDistCorr;

	}

	projData[angle*projPitch+detector] += fVal;
}

bool FanFP_internal(float* D_volumeData, unsigned int volumePitch,
           float* D_projData, unsigned int projPitch,
           const SDimensions& dims, const SFanProjection* angles,
           float outputScale)
{
	assert(dims.iProjAngles <= g_MaxAngles);

	hipArray* D_dataArray;
	hipTextureObject_t D_texObj;

	if (!createArrayAndTextureObject2D(D_volumeData, D_dataArray, D_texObj, volumePitch, dims.iVolWidth, dims.iVolHeight))
		return false;

	// transfer angles to constant memory
	float* tmp = new float[dims.iProjAngles];

#define TRANSFER_TO_CONSTANT(name) do { for (unsigned int i = 0; i < dims.iProjAngles; ++i) tmp[i] = angles[i].f##name ; hipMemcpyToSymbol(HIP_SYMBOL(gC_##name), tmp, dims.iProjAngles*sizeof(float), 0, hipMemcpyHostToDevice); } while (0)

	TRANSFER_TO_CONSTANT(SrcX);
	TRANSFER_TO_CONSTANT(SrcY);
	TRANSFER_TO_CONSTANT(DetSX);
	TRANSFER_TO_CONSTANT(DetSY);
	TRANSFER_TO_CONSTANT(DetUX);
	TRANSFER_TO_CONSTANT(DetUY);

#undef TRANSFER_TO_CONSTANT

	delete[] tmp;

	dim3 dimBlock(g_detBlockSize, g_anglesPerBlock); // region size, angles
	const unsigned int g_blockSliceSize = g_detBlockSize;

	std::list<hipStream_t> streams;


	unsigned int blockStart = 0;
	unsigned int blockEnd = dims.iProjAngles;

	dim3 dimGrid((blockEnd-blockStart+g_anglesPerBlock-1)/g_anglesPerBlock,
	             (dims.iProjDets+g_blockSliceSize-1)/g_blockSliceSize); // angle blocks, regions
	hipStream_t stream1;
	hipStreamCreate(&stream1);
	streams.push_back(stream1);
	for (unsigned int i = 0; i < dims.iVolWidth; i += g_blockSlices)
		FanFPhorizontal<<<dimGrid, dimBlock, 0, stream1>>>(D_projData, projPitch, D_texObj, i, blockStart, blockEnd, dims, outputScale);

	hipStream_t stream2;
	hipStreamCreate(&stream2);
	streams.push_back(stream2);
	for (unsigned int i = 0; i < dims.iVolHeight; i += g_blockSlices)
		FanFPvertical<<<dimGrid, dimBlock, 0, stream2>>>(D_projData, projPitch, D_texObj, i, blockStart, blockEnd, dims, outputScale);

	bool ok = true;

	ok &= checkCuda(hipStreamSynchronize(stream1), "fan_fp hor");
	hipStreamDestroy(stream1);

	ok &= checkCuda(hipStreamSynchronize(stream2), "fan_fp ver");
	hipStreamDestroy(stream2);

	hipFreeArray(D_dataArray);

	hipDestroyTextureObject(D_texObj);

	return ok;
}

bool FanFP(float* D_volumeData, unsigned int volumePitch,
           float* D_projData, unsigned int projPitch,
           const SDimensions& dims, const SFanProjection* angles,
           float outputScale)
{
	for (unsigned int iAngle = 0; iAngle < dims.iProjAngles; iAngle += g_MaxAngles) {
		SDimensions subdims = dims;
		unsigned int iEndAngle = iAngle + g_MaxAngles;
		if (iEndAngle >= dims.iProjAngles)
			iEndAngle = dims.iProjAngles;
		subdims.iProjAngles = iEndAngle - iAngle;

		bool ret;
		ret = FanFP_internal(D_volumeData, volumePitch,
		                         D_projData + iAngle * projPitch, projPitch,
		                         subdims, angles + iAngle,
		                         outputScale);
		if (!ret)
			return false;
	}
	return true;
}

}
