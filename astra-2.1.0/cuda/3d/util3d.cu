#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2022, imec Vision Lab, University of Antwerp
           2014-2022, CWI, Amsterdam

Contact: astra@astra-toolbox.com
Website: http://www.astra-toolbox.com/

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
*/

#include "../../astra/cuda/3d/util3d.h"

#include "../../astra/cuda/2d/util.h"

#include "../../astra/Logging.h"

#include <cstdio>
#include <cassert>

namespace astraCUDA3d {


hipPitchedPtr allocateVolumeData(const SDimensions3D& dims)
{
	hipExtent extentV;
	extentV.width = dims.iVolX*sizeof(float);
	extentV.height = dims.iVolY;
	extentV.depth = dims.iVolZ;

	hipPitchedPtr volData;

	if (!checkCuda(hipMalloc3D(&volData, extentV), "allocateVolumeData 3D")) {
		ASTRA_ERROR("Failed to allocate %dx%dx%d GPU buffer", dims.iVolX, dims.iVolY, dims.iVolZ);
		volData.ptr = 0;
	}

	return volData;
}
hipPitchedPtr allocateProjectionData(const SDimensions3D& dims)
{
	hipExtent extentP;
	extentP.width = dims.iProjU*sizeof(float);
	extentP.height = dims.iProjAngles;
	extentP.depth = dims.iProjV;

	hipPitchedPtr projData;

	if (!checkCuda(hipMalloc3D(&projData, extentP), "allocateProjectionData 3D")) {
		ASTRA_ERROR("Failed to allocate %dx%dx%d GPU buffer", dims.iProjU, dims.iProjAngles, dims.iProjV);
		projData.ptr = 0;
	}

	return projData;
}
bool zeroVolumeData(hipPitchedPtr& D_data, const SDimensions3D& dims)
{
	char* t = (char*)D_data.ptr;

	for (unsigned int z = 0; z < dims.iVolZ; ++z) {
		if (!checkCuda(hipMemset2D(t, D_data.pitch, 0, dims.iVolX*sizeof(float), dims.iVolY), "zeroVolumeData 3D")) {
			return false;
		}
		t += D_data.pitch * dims.iVolY;
	}
	return true;
}
bool zeroProjectionData(hipPitchedPtr& D_data, const SDimensions3D& dims)
{
	char* t = (char*)D_data.ptr;

	for (unsigned int z = 0; z < dims.iProjV; ++z) {
		if (!checkCuda(hipMemset2D(t, D_data.pitch, 0, dims.iProjU*sizeof(float), dims.iProjAngles), "zeroProjectionData 3D")) {
			return false;
		}
		t += D_data.pitch * dims.iProjAngles;
	}

	return true;
}
bool copyVolumeToDevice(const float* data, hipPitchedPtr& D_data, const SDimensions3D& dims, unsigned int pitch)
{
	if (!pitch)
		pitch = dims.iVolX;

	hipPitchedPtr ptr;
	ptr.ptr = (void*)data; // const cast away
	ptr.pitch = pitch*sizeof(float);
	ptr.xsize = dims.iVolX*sizeof(float);
	ptr.ysize = dims.iVolY;

	hipExtent extentV;
	extentV.width = dims.iVolX*sizeof(float);
	extentV.height = dims.iVolY;
	extentV.depth = dims.iVolZ;

	hipPos zp = { 0, 0, 0 };

	hipMemcpy3DParms p;
	p.srcArray = 0;
	p.srcPos = zp;
	p.srcPtr = ptr;
	p.dstArray = 0;
	p.dstPos = zp;
	p.dstPtr = D_data;
	p.extent = extentV;
	p.kind = hipMemcpyHostToDevice;

	return checkCuda(hipMemcpy3D(&p), "copyVolumeToDevice 3D");
}

bool copyProjectionsToDevice(const float* data, hipPitchedPtr& D_data, const SDimensions3D& dims, unsigned int pitch)
{
	if (!pitch)
		pitch = dims.iProjU;

	hipPitchedPtr ptr;
	ptr.ptr = (void*)data; // const cast away
	ptr.pitch = pitch*sizeof(float);
	ptr.xsize = dims.iProjU*sizeof(float);
	ptr.ysize = dims.iProjAngles;

	hipExtent extentV;
	extentV.width = dims.iProjU*sizeof(float);
	extentV.height = dims.iProjAngles;
	extentV.depth = dims.iProjV;

	hipPos zp = { 0, 0, 0 };

	hipMemcpy3DParms p;
	p.srcArray = 0;
	p.srcPos = zp;
	p.srcPtr = ptr;
	p.dstArray = 0;
	p.dstPos = zp;
	p.dstPtr = D_data;
	p.extent = extentV;
	p.kind = hipMemcpyHostToDevice;

	return checkCuda(hipMemcpy3D(&p), "copyProjectionsToDevice 3D");
}

bool copyVolumeFromDevice(float* data, const hipPitchedPtr& D_data, const SDimensions3D& dims, unsigned int pitch)
{
	if (!pitch)
		pitch = dims.iVolX;

	hipPitchedPtr ptr;
	ptr.ptr = data;
	ptr.pitch = pitch*sizeof(float);
	ptr.xsize = dims.iVolX*sizeof(float);
	ptr.ysize = dims.iVolY;

	hipExtent extentV;
	extentV.width = dims.iVolX*sizeof(float);
	extentV.height = dims.iVolY;
	extentV.depth = dims.iVolZ;

	hipPos zp = { 0, 0, 0 };

	hipMemcpy3DParms p;
	p.srcArray = 0;
	p.srcPos = zp;
	p.srcPtr = D_data;
	p.dstArray = 0;
	p.dstPos = zp;
	p.dstPtr = ptr;
	p.extent = extentV;
	p.kind = hipMemcpyDeviceToHost;

	return checkCuda(hipMemcpy3D(&p), "copyVolumeFromDevice 3D");
}

bool copyProjectionsFromDevice(float* data, const hipPitchedPtr& D_data, const SDimensions3D& dims, unsigned int pitch)
{
	if (!pitch)
		pitch = dims.iProjU;

	hipPitchedPtr ptr;
	ptr.ptr = data;
	ptr.pitch = pitch*sizeof(float);
	ptr.xsize = dims.iProjU*sizeof(float);
	ptr.ysize = dims.iProjAngles;

	hipExtent extentV;
	extentV.width = dims.iProjU*sizeof(float);
	extentV.height = dims.iProjAngles;
	extentV.depth = dims.iProjV;

	hipPos zp = { 0, 0, 0 };

	hipMemcpy3DParms p;
	p.srcArray = 0;
	p.srcPos = zp;
	p.srcPtr = D_data;
	p.dstArray = 0;
	p.dstPos = zp;
	p.dstPtr = ptr;
	p.extent = extentV;
	p.kind = hipMemcpyDeviceToHost;

	return checkCuda(hipMemcpy3D(&p), "copyProjectionsFromDevice 3D");
}

bool duplicateVolumeData(hipPitchedPtr& D_dst, const hipPitchedPtr& D_src, const SDimensions3D& dims)
{
	hipExtent extentV;
	extentV.width = dims.iVolX*sizeof(float);
	extentV.height = dims.iVolY;
	extentV.depth = dims.iVolZ;

	hipPos zp = { 0, 0, 0 };

	hipMemcpy3DParms p;
	p.srcArray = 0;
	p.srcPos = zp;
	p.srcPtr = D_src;
	p.dstArray = 0;
	p.dstPos = zp;
	p.dstPtr = D_dst;
	p.extent = extentV;
	p.kind = hipMemcpyDeviceToDevice;

	return checkCuda(hipMemcpy3D(&p), "duplicateVolumeData 3D");
}

bool duplicateProjectionData(hipPitchedPtr& D_dst, const hipPitchedPtr& D_src, const SDimensions3D& dims)
{
	hipExtent extentV;
	extentV.width = dims.iProjU*sizeof(float);
	extentV.height = dims.iProjAngles;
	extentV.depth = dims.iProjV;

	hipPos zp = { 0, 0, 0 };

	hipMemcpy3DParms p;
	p.srcArray = 0;
	p.srcPos = zp;
	p.srcPtr = D_src;
	p.dstArray = 0;
	p.dstPos = zp;
	p.dstPtr = D_dst;
	p.extent = extentV;
	p.kind = hipMemcpyDeviceToDevice;

	return checkCuda(hipMemcpy3D(&p), "duplicateProjectionData 3D");
}



// TODO: Consider using a single array of size max(proj,volume) (per dim)
//       instead of allocating a new one each time

hipArray* allocateVolumeArray(const SDimensions3D& dims)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipArray* cuArray;
	hipExtent extentA;
	extentA.width = dims.iVolX;
	extentA.height = dims.iVolY;
	extentA.depth = dims.iVolZ;

	if (!checkCuda(hipMalloc3DArray(&cuArray, &channelDesc, extentA), "allocateVolumeArray 3D")) {
		ASTRA_ERROR("Failed to allocate %dx%dx%d GPU array", dims.iVolX, dims.iVolY, dims.iVolZ);
		return 0;
	}

	return cuArray;
}
hipArray* allocateProjectionArray(const SDimensions3D& dims)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipArray* cuArray;
	hipExtent extentA;
	extentA.width = dims.iProjU;
	extentA.height = dims.iProjAngles;
	extentA.depth = dims.iProjV;

	if (!checkCuda(hipMalloc3DArray(&cuArray, &channelDesc, extentA), "allocateProjectionArray 3D")) {
		ASTRA_ERROR("Failed to allocate %dx%dx%d GPU array", dims.iProjU, dims.iProjAngles, dims.iProjV);
		return 0;
	}

	return cuArray;
}

bool transferVolumeToArray(hipPitchedPtr D_volumeData, hipArray* array, const SDimensions3D& dims)
{
	hipExtent extentA;
	extentA.width = dims.iVolX;
	extentA.height = dims.iVolY;
	extentA.depth = dims.iVolZ;

	hipMemcpy3DParms p;
	hipPos zp = {0, 0, 0};
	p.srcArray = 0;
	p.srcPos = zp;
	p.srcPtr = D_volumeData;
	p.dstArray = array;
	p.dstPtr.ptr = 0;
	p.dstPtr.pitch = 0;
	p.dstPtr.xsize = 0;
	p.dstPtr.ysize = 0;
	p.dstPos = zp;
	p.extent = extentA;
	p.kind = hipMemcpyDeviceToDevice;

	return checkCuda(hipMemcpy3D(&p), "transferVolumeToArray 3D");
}

bool transferProjectionsToArray(hipPitchedPtr D_projData, hipArray* array, const SDimensions3D& dims)
{
	hipExtent extentA;
	extentA.width = dims.iProjU;
	extentA.height = dims.iProjAngles;
	extentA.depth = dims.iProjV;

	hipMemcpy3DParms p;
	hipPos zp = {0, 0, 0};
	p.srcArray = 0;
	p.srcPos = zp;
	p.srcPtr = D_projData;
	p.dstArray = array;
	p.dstPtr.ptr = 0;
	p.dstPtr.pitch = 0;
	p.dstPtr.xsize = 0;
	p.dstPtr.ysize = 0;
	p.dstPos = zp;
	p.extent = extentA;
	p.kind = hipMemcpyDeviceToDevice;

	return checkCuda(hipMemcpy3D(&p), "transferProjectionsToArray 3D");
}

bool transferHostProjectionsToArray(const float *projData, hipArray* array, const SDimensions3D& dims)
{
	hipExtent extentA;
	extentA.width = dims.iProjU;
	extentA.height = dims.iProjAngles;
	extentA.depth = dims.iProjV;

	hipPitchedPtr ptr;
	ptr.ptr = (void*)projData; // const cast away
	ptr.pitch = dims.iProjU*sizeof(float);
	ptr.xsize = dims.iProjU*sizeof(float);
	ptr.ysize = dims.iProjAngles;

	hipMemcpy3DParms p;
	hipPos zp = {0, 0, 0};
	p.srcArray = 0;
	p.srcPos = zp;
	p.srcPtr = ptr;
	p.dstArray = array;
	p.dstPtr.ptr = 0;
	p.dstPtr.pitch = 0;
	p.dstPtr.xsize = 0;
	p.dstPtr.ysize = 0;
	p.dstPos = zp;
	p.extent = extentA;
	p.kind = hipMemcpyHostToDevice;

	return checkCuda(hipMemcpy3D(&p), "transferHostProjectionsToArray 3D");
}

bool createTextureObject3D(hipArray* array, hipTextureObject_t& texObj)
{
	hipChannelFormatDesc channelDesc =
	    hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = array;

	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0] = hipAddressModeBorder;
	texDesc.addressMode[1] = hipAddressModeBorder;
	texDesc.addressMode[2] = hipAddressModeBorder;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 0;

	return checkCuda(hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL), "createTextureObject3D");
}



float dotProduct3D(hipPitchedPtr data, unsigned int x, unsigned int y,
                   unsigned int z)
{
	return astraCUDA::dotProduct2D((float*)data.ptr, data.pitch/sizeof(float), x, y*z);
}


int calcNextPowerOfTwo(int _iValue)
{
	int iOutput = 1;
	while(iOutput < _iValue)
		iOutput *= 2;
	return iOutput;
}

}
